#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <>
#include <stdlib.h>

#define THREADS_PER_BLOCK 16
// saves matrix to file
void save_to_file(double *AB, const int a_size) {
    FILE *f = fopen("out.txt", "w+");

    fprintf(f, "%d\n", a_size);

    for(int i = 0; i < a_size*(a_size + 1); i++) {
        if((i + 1) % (a_size+1) == 0) {
            fprintf(f, "\n");
            continue;
        }
        fprintf(f, "%lf ", AB[i]);
    }

    for(int i = 0; i < a_size; i++)
        fprintf(f, "%lf ", AB[a_size + i*(a_size + 1)]);
    fprintf(f, "\n");

    fclose(f);
}
// allocates matrix
double *alloc_matrix(const int a, const int b) {
    return (double*)malloc(sizeof(double *)*a*b);
}
//reads matrix from file
double *load_from_file(int *a_size, char *name) {
    FILE *f = fopen(name, "r");

    int size;
    fscanf(f, "%d", &size);
    double *matrix_ab = alloc_matrix(size, size + 1);

    for(int i = 0; i < size*(size + 1); i++) {
        if((i+1) % (size + 1) == 0)
            continue;
        fscanf(f, "%lf", &matrix_ab[i]);
    }

    for(int i = 0; i < size; i++) {
        fscanf(f, "%lf", &matrix_ab[size + i*(size + 1)]);
    }

    fclose(f);

    *a_size = size;

    return matrix_ab;
}


// prints matrix to terminal
void print_output(double *AB, const int a_size) {
    printf("%d\n", a_size);

    for(int i = 0; i < a_size*(a_size + 1); i++) {
        if((i + 1) % (a_size+1) == 0) {
            printf("\n");
            continue;
        }
        printf("%lf\t", AB[i]);
    }

    for(int i = 0; i < a_size; i++)
        printf("%lf\t", AB[a_size + i*(a_size + 1)]);
    printf("\n");
}
// reads size from terminal
int load_size() {
    int size;
    scanf("%d", &size);
    return size;
}
// reads matrix from terminal
void load_ab_matrix(double *a, const int size) {
    for(int i = 0; i < size*(size + 1); i++) {
        if((i+1) % (size + 1) == 0)
            continue;
        scanf("%lf", &a[i]);
    }

    for(int i = 0; i < size; i++) {
        scanf("%lf", &a[size + i*(size + 1)]);
    }
}

double  *load_input(int *size) {
    *size = load_size();
    double *matrix_ab = alloc_matrix(*size, *size + 1);
    load_ab_matrix(matrix_ab, *size);
    return matrix_ab;
}

// replaces zeros in matrix
__global__ void replace_zero_gpu(double *AB, int rows, int columns, int column) {
    if(fabs(AB[column*columns + column]) <= 1e-4) {
        int row = column;
        for(; row < rows; row++) {
            if(fabs(AB[row*columns + column]) > 1e-4)
                break;
        }
        int threadId = blockDim.x*blockIdx.x + threadIdx.x;
        if(threadId + column >= columns)
            return;

        int zero = column*columns + column + threadId;
        int chosen = row*columns + column + threadId;
        AB[zero] += AB[chosen];
    }
}

//eliminates columns
__global__ void column_elimination_gpu(double *AB, int rows, int columns, int column) {
    int threadId = blockDim.x*blockIdx.x + threadIdx.x;
    if(threadId >= (rows - 1 - column)*(columns - column))
        return;

    int el_row = column + threadId/(columns - column) + 1;
    int el_col = column + threadId%(columns - column);
    int el = el_col + el_row*columns;
    int upper_el = el_col + column*columns;

    int main_el = column + column*columns;
    int main2_el = column + el_row*columns;
    double f = AB[main2_el]/AB[main_el];

    AB[el] -= f*AB[upper_el];
}

//multiplies columns
__global__ void multiple_column(double *AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;
    AB[(threadId * columns) + row] *= AB[columns*(row + 1) - 1];
}

// eliminate rows from bottom
__global__ void reverse_row_elimination(double *AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;
    int cols = columns - 2 - row;

    int start_index = row*columns + row + 1;

    int j = cols%2;
    for(int i = cols/2; i > 0; i/=2) {
        if(threadId >= i)
            return;

        AB[start_index + threadId] += (AB[start_index + threadId + i + j]);
        AB[start_index + threadId + i + j] = 0;
        if(j == 1)
            i++;
        j = i%2;
        __syncthreads();
    }

    int x_el = (row + 1)*columns - 1;
    int diag_el = row*columns + row;

    if(diag_el + 1 != x_el) {
        AB[x_el] -= AB[diag_el + 1];
        AB[diag_el + 1] = 0.0;
    }

    AB[x_el] /= AB[diag_el];
    AB[diag_el] = 1.0;
}

// sum rows
__global__ void sum_row(double *AB, int rows, int columns, int row) {
    int threadId = threadIdx.x;

    int j = columns%2;
    for(int i = columns/2; i > 0; i/=2) {
        if(threadId >= i)
            return;

        AB[threadId] += AB[threadId + i + j];
        __syncthreads();
        if(j == 1)
            i++;
        j = i%2;
    }
}

// perform a gaussian elimination
void start_gaussian_elimination_gpu(double *AB, int rows, int cols) {
    double *AB_gpu;

    hipMalloc(&AB_gpu, sizeof(double)*rows*cols);
    hipMemcpy(AB_gpu, (void*)AB, sizeof(double)*rows*cols, hipMemcpyHostToDevice);

    int block_size;

    for(int column = 0; column < cols - 1; column++) {
        block_size = (cols - column - 1)/THREADS_PER_BLOCK + 1;
        replace_zero_gpu<<<block_size, THREADS_PER_BLOCK>>>(AB_gpu, rows, cols, column);
        hipDeviceSynchronize();

        block_size = ((rows - column)*(cols - column) - 1)/THREADS_PER_BLOCK + 1;
        column_elimination_gpu<<<block_size, THREADS_PER_BLOCK>>>(AB_gpu, rows, cols, column);
        hipDeviceSynchronize();
    }

    for(int row = rows - 1; row >= 0; row--) {
        reverse_row_elimination<<<1, cols>>>(AB_gpu, rows, cols, row);
        multiple_column<<<1, row>>>(AB_gpu, rows, cols, row);

        hipDeviceSynchronize();
    }

    hipMemcpy(AB, (void*)AB_gpu, sizeof(double)*rows*cols, hipMemcpyDeviceToHost);

    hipFree(AB_gpu);
}


int main(int argc, char ** argv) {
    int size;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    double *AB = load_from_file(&size, argv[1]);

    print_output(AB, size);

    hipEventRecord(start);

    start_gaussian_elimination_gpu(AB, size, size + 1);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("\n\n");

    print_output(AB, size);
    printf("\nExecution time is: %f", time);

    save_to_file(AB, size);
    FILE *f = fopen("out.txt", "w+");
    fprintf(f, "\nExecution time is: %f", time);

    return 0;
}